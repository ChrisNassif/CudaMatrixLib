#include <hip/hip_runtime.h>

void move_float_array_from_cpu_to_gpu(float* cpu_data_array, int array_size, float* gpu_data_array) {
    hipMalloc((void**) &cpu_data_array, sizeof(float) * array_size);
    hipMemcpy(gpu_data_array, cpu_data_array, sizeof(float) * array_size, hipMemcpyHostToDevice);
}

void move_float_array_from_gpu_to_cpu(float* gpu_data_array, int array_size, float* cpu_data_array) {
    hipMemcpy(cpu_data_array, gpu_data_array, sizeof(float) * array_size, hipMemcpyDeviceToHost);
    hipFree(gpu_data_array);      
}