#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_matrix_lib.h"
#include <chrono>

#define OPERATIONS_PER_THREAD 1024

__global__ void hadamard_product_kernel(float* d_vector1, float* d_vector2, float* d_output_vector, int* vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;

    for (int index = 0; index < OPERATIONS_PER_THREAD; index++) {
        int input_vector_index = thread_start_index + index;

        if (input_vector_index >= *vector_size) return;

        d_output_vector[input_vector_index] = d_vector1[input_vector_index] * d_vector2[input_vector_index];
    }
}


std::vector<float> CudaMatrixLib::hadamard_product(std::vector<float> vector1, std::vector<float> vector2) {

    int device_count;
    hipGetDeviceCount(&device_count);

    std::cout << device_count << std::endl;

    // auto program_start_time = std::chrono::high_resolution_clock::now();

    if (vector1.size() != vector2.size()) {
        return std::vector<float>();
    }
    
    int vector_size = vector1.size();
    int* h_vector_size = &vector_size;
    int* d_vector_size;

    float* h_vector1 = &vector1[0]; float* h_vector2 = &vector2[0];
    float* h_output_vector = (float*) malloc(sizeof(float) * vector_size);

    float* d_vector1; float* d_vector2; float* d_output_vector;


    const int thread_count = min(vector_size/ OPERATIONS_PER_THREAD + 1, 1024);
    const int block_count = vector_size / OPERATIONS_PER_THREAD / 1024 + 1;
    

    std::cout << "Cuda Thread Count: " << thread_count << std::endl;
    std::cout << "Cuda Block Count: " << block_count << std::endl;


    hipMalloc((void**) &d_vector1, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector2, sizeof(float) * vector_size);
    hipMalloc((void**) &d_output_vector, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector_size, sizeof(int));

    hipMemcpy(d_vector1, h_vector1, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector2, h_vector2, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_size, h_vector_size, sizeof(int), hipMemcpyHostToDevice);
    
    auto program_start_time = std::chrono::high_resolution_clock::now();

    hadamard_product_kernel <<<block_count, thread_count>>> (d_vector1, d_vector2, d_output_vector, d_vector_size);
    hipDeviceSynchronize();

    auto program_end_time = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_output_vector, d_output_vector, sizeof(float) * vector_size, hipMemcpyDeviceToHost);

    hipFree(d_vector1);      
    hipFree(d_vector2);        
    hipFree(d_output_vector);  
    hipFree(d_vector_size);

    std::vector<float> result;
    result.insert(result.end(), h_output_vector, h_output_vector + vector_size);

    
    // auto program_end_time = std::chrono::high_resolution_clock::now();
    float program_duration = std::chrono::duration_cast<std::chrono::microseconds>(program_end_time - program_start_time).count();
    std::cout << "Time (microseconds) in cuda hadamard product program: " << program_duration << std::endl;

    return result;
}

