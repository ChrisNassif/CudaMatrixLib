#include "hip/hip_runtime.h"
#include "cuda_matrix_lib.h"

__global__ void vector_sum_kernel(float* d_vector, float* d_output, int* vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    // if (thread_id >= *vector_size) return;

    // d_output_vector[thread_id] = d_vector1[thread_id] + d_vector2[thread_id];

}


float CudaMatrixLib::vector_sum(std::vector<float> vector) {    

    int vector_size = vector.size();
    int* h_vector_size = &vector_size;
    int* d_vector_size;

    float* h_vector = &vector[0];
    float* h_output = (float*) malloc(sizeof(float) * vector_size);

    float* d_vector; float* d_output;


    const int threadCount = min(vector_size, 1024);
    const int blockCount = vector_size / 1024 + 1;


    hipMalloc((void**) &d_vector, sizeof(float) * vector_size);
    hipMalloc((void**) &d_output, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector_size, sizeof(int));

    hipMemcpy(d_vector, h_vector, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_size, h_vector_size, sizeof(int), hipMemcpyHostToDevice);
    
    vector_sum_kernel <<<blockCount, threadCount>>> (d_vector, d_output, d_vector_size);

    hipMemcpy(h_output, d_output, sizeof(float) * vector_size, hipMemcpyDeviceToHost);

    hipFree(d_vector);       
    hipFree(d_output);  
    hipFree(d_vector_size);


    return *h_output;
}
