#include "hip/hip_runtime.h"
#include "cuda_matrix_lib.h"
#include "cuda_kernels.h"

float CudaMatrixLib::vector_sum(std::vector<float> input_vector) {    

    
    int input_array_size = input_vector.size();
    int output_array_size = input_array_size/ OPERATIONS_PER_THREAD + 1;

    int* h_input_array_size = &(input_array_size);
    float* h_input_array = &input_vector[0]; 
    float* h_output_array = (float*) malloc(sizeof(float) * input_array_size);

    int* d_input_array_size;
    float* d_input_array; 
    float* d_output_array;


    const int thread_count = min(output_array_size, 1024);
    const int block_count = input_array_size / OPERATIONS_PER_THREAD / 1024 + 1;


    hipMalloc((void**) &d_input_array, sizeof(float) * input_array_size);
    hipMalloc((void**) &d_output_array, sizeof(float) * input_array_size);
    hipMalloc((void**) &d_input_array_size, sizeof(int));

    hipMemcpy(d_input_array, h_input_array, sizeof(float) * input_array_size, hipMemcpyHostToDevice);
    hipMemcpy(d_input_array_size, h_input_array_size, sizeof(int), hipMemcpyHostToDevice);
    
    auto program_start_time = std::chrono::high_resolution_clock::now();

    vector_sum_kernel <<<block_count, thread_count>>> (d_input_array, d_output_array, d_input_array_size);
    hipDeviceSynchronize();

    auto program_end_time = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_output_array, d_output_array, sizeof(float) * output_array_size, hipMemcpyDeviceToHost);

    hipFree(d_input_array_size);
    hipFree(d_input_array);      
    hipFree(d_output_array);  

    float result = 0;
    for (int i = 0; i < output_array_size; i++) {
        result += h_output_array[i];
    }

    float program_duration = std::chrono::duration_cast<std::chrono::microseconds>(program_end_time - program_start_time).count();
    std::cout << "Time (microseconds) in cuda vector sum program: " << program_duration << std::endl;

    return result;
}