#include "hip/hip_runtime.h"
#include "cuda_matrix_lib.h"
// #include "../utils/helper_function_library.hpp"

#define OPERATIONS_PER_THREAD 1024


__global__ void vector_sum_kernel(float* d_vector, float* d_output, int* input_vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;
    
    for (int operation_index = 0; operation_index < OPERATIONS_PER_THREAD; operation_index++) {
        int vector_index = thread_start_index + operation_index;

        if (vector_index >= *input_vector_size) return;

        d_output[thread_id] += d_vector[vector_index];
    }

}


float CudaMatrixLib::vector_sum(std::vector<float> vector) {    

    int vector_size = vector.size();
    int* h_vector_size = &vector_size;
    int* d_vector_size;

    float* h_vector = &vector[0];
    int output_vector_size = vector_size/ OPERATIONS_PER_THREAD + 1;
    float* h_output_vector = (float*) malloc(sizeof(float) * output_vector_size);

    float* d_vector; 
    float* d_output_vector;

    const int thread_count = min(vector_size/ OPERATIONS_PER_THREAD + 1, 1024);
    const int block_count = vector_size / OPERATIONS_PER_THREAD / 1024 + 1;

    std::cout << thread_count << std::endl;
    std::cout << block_count << std::endl;

    hipMalloc((void**) &d_vector, sizeof(float) * vector_size);
    hipMalloc((void**) &d_output_vector, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector_size, sizeof(int));

    hipMemcpy(d_vector, h_vector, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_size, h_vector_size, sizeof(int), hipMemcpyHostToDevice);
    
    auto program_start_time = std::chrono::high_resolution_clock::now();

    std::cout << "hi1" << std::endl;
    vector_sum_kernel <<<block_count, thread_count>>> (d_vector, d_output_vector, d_vector_size);
    hipDeviceSynchronize();
    std::cout << "hi2" << std::endl;

    auto program_end_time = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_output_vector, d_output_vector, sizeof(float) * output_vector_size, hipMemcpyDeviceToHost);

    hipFree(d_vector);       
    hipFree(d_output_vector);  
    hipFree(d_vector_size);

    std::vector<float> result1;
    result1.insert(result1.end(), h_output_vector, h_output_vector + output_vector_size); 
    // print_vector(result1);


    // sum up the result
    float result = 0;
    for (int i = 0; i < output_vector_size; i++) {
        result += h_output_vector[i];
    }


    float program_duration = std::chrono::duration_cast<std::chrono::microseconds>(program_end_time - program_start_time).count();
    std::cout << "Time (microseconds) in cuda vector sum program: " << program_duration << std::endl;

    return result;
}
