#include "hip/hip_runtime.h"
#include "cuda_matrix_lib.h"
#include "cuda_kernels.h"


__device__ __host__ float elementwise_function(float input) {
    return -1 * input;
}

__device__ pointerToElementwiseFunction_t d_pointer_to_elementwise_function = elementwise_function;



std::vector<float> CudaMatrixLib::test(std::vector<float> input_vector) {    
    
    int input_array_size = input_vector.size();
    int output_array_size = input_array_size;

    int* h_input_array_size = &(input_array_size);
    float* h_input_array = &input_vector[0]; 
    float* h_output_array = (float*) malloc(sizeof(float) * input_array_size);

    int* d_input_array_size;
    float* d_input_array; 
    float* d_output_array;
    
    pointerToElementwiseFunction_t h_pointer_to_elementwise_function;


    const int thread_count = min(input_array_size/ OPERATIONS_PER_THREAD + 1, 1024);
    const int block_count = input_array_size / OPERATIONS_PER_THREAD / 1024 + 1;


    hipMalloc((void**) &d_input_array, sizeof(float) * input_array_size);
    hipMalloc((void**) &d_output_array, sizeof(float) * input_array_size);
    hipMalloc((void**) &d_input_array_size, sizeof(int));

    hipMemcpy(d_input_array, h_input_array, sizeof(float) * input_array_size, hipMemcpyHostToDevice);
    hipMemcpy(d_input_array_size, h_input_array_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyFromSymbol(&h_pointer_to_elementwise_function, HIP_SYMBOL(d_pointer_to_elementwise_function), sizeof(pointerToElementwiseFunction_t), 0, hipMemcpyDeviceToHost);
    
    auto program_start_time = std::chrono::high_resolution_clock::now();
    
    apply_elementwise_function_kernel <<<block_count, thread_count>>> (d_input_array, h_pointer_to_elementwise_function, d_output_array, d_input_array_size);
    hipDeviceSynchronize();

    auto program_end_time = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_output_array, d_output_array, sizeof(float) * output_array_size, hipMemcpyDeviceToHost);

    hipFree(d_input_array_size);
    hipFree(d_input_array);
    hipFree(d_output_array);  


    std::vector<float> result = c_array_to_std_vector(h_output_array, output_array_size);

    float program_duration = std::chrono::duration_cast<std::chrono::microseconds>(program_end_time - program_start_time).count();
    std::cout << "Time (microseconds) in cuda scalar multiplication program: " << program_duration << std::endl;

    return result;
}