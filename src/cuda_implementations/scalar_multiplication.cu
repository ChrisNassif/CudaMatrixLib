#include "hip/hip_runtime.h"
#include "cuda_matrix_lib.h"
#include "cuda_kernels.h"


std::vector<float> CudaMatrixLib::scalar_multiplication(std::vector<float> input_vector, float input_scalar) {    
    
    int input_array_size = input_vector.size();
    int output_array_size = input_array_size;

    int* h_input_array_size = &(input_array_size);
    float* h_input_array = &input_vector[0]; 
    float* h_input_scalar = &input_scalar;
    float* h_output_array = (float*) malloc(sizeof(float) * input_array_size);

    int* d_input_array_size;
    float* d_input_array; 
    float* d_input_scalar;
    float* d_output_array;


    const int thread_count = min(input_array_size/ OPERATIONS_PER_THREAD + 1, 1024);
    const int block_count = input_array_size / OPERATIONS_PER_THREAD / 1024 + 1;


    hipMalloc((void**) &d_input_array, sizeof(float) * input_array_size);
    hipMalloc((void**) &d_input_scalar, sizeof(float));
    hipMalloc((void**) &d_output_array, sizeof(float) * input_array_size);
    hipMalloc((void**) &d_input_array_size, sizeof(int));

    hipMemcpy(d_input_array, h_input_array, sizeof(float) * input_array_size, hipMemcpyHostToDevice);
    hipMemcpy(d_input_scalar, h_input_scalar, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input_array_size, h_input_array_size, sizeof(int), hipMemcpyHostToDevice);
    
    auto program_start_time = std::chrono::high_resolution_clock::now();

    scalar_multiplication_kernel <<<block_count, thread_count>>> (d_input_array, d_input_scalar, d_output_array, d_input_array_size);
    hipDeviceSynchronize();

    auto program_end_time = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_output_array, d_output_array, sizeof(float) * output_array_size, hipMemcpyDeviceToHost);

    hipFree(d_input_array_size);
    hipFree(d_input_array);
    hipFree(d_input_scalar);
    hipFree(d_output_array);  


    std::vector<float> result = c_array_to_std_vector(h_output_array, output_array_size);

    float program_duration = std::chrono::duration_cast<std::chrono::microseconds>(program_end_time - program_start_time).count();
    std::cout << "Time (microseconds) in cuda scalar multiplication program: " << program_duration << std::endl;

    return result;
}
