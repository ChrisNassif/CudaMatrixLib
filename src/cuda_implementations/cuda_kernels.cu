#include "hip/hip_runtime.h"
#include "cuda_kernels.h"


__global__ void hadamard_product_kernel(float* d_input_vector1, float* d_input_vector2, float* d_output_vector, int* d_input_vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;

    for (int operation_index = 0; operation_index < OPERATIONS_PER_THREAD; operation_index++) {
        int vector_index = thread_start_index + operation_index;

        if (vector_index >= *d_input_vector_size) return;

        d_output_vector[vector_index] = d_input_vector1[vector_index] * d_input_vector2[vector_index];
    }
}


__global__ void scalar_multiplication_kernel(float* d_input_vector, float* d_input_scalar, float* d_output_vector, int* d_input_vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;
    
    for (int operation_index = 0; operation_index < OPERATIONS_PER_THREAD; operation_index++) {
        int vector_index = thread_start_index + operation_index;

        if (vector_index >= *d_input_vector_size) return;

        d_output_vector[vector_index] *= (*d_input_scalar);
    }
}


__global__ void vector_addition_kernel(float* d_input_vector1, float* d_input_vector2, float* d_output_vector, int* d_input_vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;

    for (int operation_index = 0; operation_index < OPERATIONS_PER_THREAD; operation_index++) {
        int vector_index = thread_start_index + operation_index;

        if (vector_index >= *d_input_vector_size) return;

        d_output_vector[vector_index] = d_input_vector1[vector_index] + d_input_vector2[vector_index];
    }
}


__global__ void vector_sum_kernel(float* d_input_vector, float* d_output_vector, int* d_input_vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;
    
    for (int operation_index = 0; operation_index < OPERATIONS_PER_THREAD; operation_index++) {
        int vector_index = thread_start_index + operation_index;

        if (vector_index >= *d_input_vector_size) return;

        d_output_vector[thread_id] += d_input_vector[vector_index];
    }
}

typedef float(*pointFunction_t)(float);

__global__ void apply_elementwise_function_kernel(float* d_input_vector, pointFunction_t d_function_to_apply, float* d_output_vector, int* d_input_vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;
    
    for (int operation_index = 0; operation_index < OPERATIONS_PER_THREAD; operation_index++) {
        int vector_index = thread_start_index + operation_index;

        if (vector_index >= *d_input_vector_size) return;

        d_output_vector[vector_index] = (*d_function_to_apply)(d_input_vector[vector_index]);
    }
}