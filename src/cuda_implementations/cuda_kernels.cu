#include "hip/hip_runtime.h"
#include "cuda_kernels.h"


__global__ void hadamard_product_kernel(float* d_input_vector1, float* d_input_vector2, float* d_output_vector, int* d_input_vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;

    for (int operation_index = 0; operation_index < OPERATIONS_PER_THREAD; operation_index++) {
        int vector_index = thread_start_index + operation_index;

        if (vector_index >= *d_input_vector_size) return;

        d_output_vector[vector_index] = d_input_vector1[vector_index] * d_input_vector2[vector_index];
    }
}


__global__ void scalar_multiplication_kernel(float* d_input_vector, float* d_input_scalar, float* d_output_vector, int* d_input_vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;
    
    for (int operation_index = 0; operation_index < OPERATIONS_PER_THREAD; operation_index++) {
        int vector_index = thread_start_index + operation_index;

        if (vector_index >= *d_input_vector_size) return;

        d_output_vector[vector_index] *= (*d_input_scalar);
    }
}


__global__ void vector_addition_kernel(float* d_input_vector1, float* d_input_vector2, float* d_output_vector, int* d_input_vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;

    for (int operation_index = 0; operation_index < OPERATIONS_PER_THREAD; operation_index++) {
        int vector_index = thread_start_index + operation_index;

        if (vector_index >= *d_input_vector_size) return;

        d_output_vector[vector_index] = d_input_vector1[vector_index] + d_input_vector2[vector_index];
    }
}


__global__ void vector_sum_kernel(float* d_input_vector, float* d_output_vector, int* d_input_vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;
    
    for (int operation_index = 0; operation_index < OPERATIONS_PER_THREAD; operation_index++) {
        int vector_index = thread_start_index + operation_index;

        if (vector_index >= *d_input_vector_size) return;

        d_output_vector[thread_id] += d_input_vector[vector_index];
    }
}


__global__ void apply_elementwise_function_kernel(float* d_input_vector, pointerToElementwiseFunction_t d_function_to_apply, float* d_output_vector, int* d_input_vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;
    
    for (int operation_index = 0; operation_index < OPERATIONS_PER_THREAD; operation_index++) {
        int vector_index = thread_start_index + operation_index;

        if (vector_index >= *d_input_vector_size) return;

        d_output_vector[vector_index] = (*d_function_to_apply)(d_input_vector[vector_index]);
    }
}


__global__ void naive_matrix_multiplication_kernel(
    float* d_input_matrix1, float* d_input_matrix2, float* d_output_matrix, 
    int* d_input_matrix1_number_of_rows_M, int* d_input_matrix1_number_of_columns_K, 
    int* d_input_matrix2_number_of_columns_N
) {
    const int output_matrix_row = threadIdx.y + blockIdx.y * blockDim.y;
    const int output_matrix_column = threadIdx.x + blockIdx.x * blockDim.x;

    const int M = (*d_input_matrix1_number_of_rows_M);
    const int K = (*d_input_matrix1_number_of_columns_K);
    const int N = (*d_input_matrix2_number_of_columns_N);

    if (output_matrix_row >= M || output_matrix_column >= N) return;

    float temp_value = 0;
    // printf("\n");
    for (int i = 0; i < K; i++) {
        // printf("(%i, %i): %i: %i\n", output_matrix_row, output_matrix_column, output_matrix_row*K + i, d_input_matrix1[output_matrix_row*K + i]);
        // printf("(%i, %i): %i: %i\n", output_matrix_row, output_matrix_column, output_matrix_column + i * N, d_input_matrix2[output_matrix_column + i * N]);
        // printf("\n");
        temp_value += d_input_matrix1[output_matrix_row*K + i] * d_input_matrix2[output_matrix_column + i * N];
    }
    // printf("\n");
    // printf("%f\n", temp_value);
    // printf("\n");
    d_output_matrix[output_matrix_row*M + output_matrix_column] = temp_value;

    // const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;
    
    // for (int operation_index = 0; operation_index < OPERATIONS_PER_THREAD; operation_index++) {
    //     int vector_index = thread_start_index + operation_index;

    //     int total_number_of_elements
    //     if (vector_index >= (*d_input_matrix1_number_of_rows_M) * (*d_input_matrix2_number_of_columns_N)) return;

    //     d_output_vector[vector_index] = (*d_function_to_apply)(d_input_vector[vector_index]);
    // } 
}