#include "hip/hip_runtime.h"
#include "cuda_matrix_lib.h"
#include "cuda_kernels.h"


std::vector<float> CudaMatrixLib::vector_addition(std::vector<float> input_vector1, std::vector<float> input_vector2) {

    if (input_vector1.size() != input_vector2.size()) {
        return std::vector<float>();
    }
    
    int input_array_size = input_vector1.size();
    int output_array_size = input_array_size;

    int* h_input_array_size = &(input_array_size);
    float* h_input_array1 = &input_vector1[0]; 
    float* h_input_array2 = &input_vector2[0];
    float* h_output_array = (float*) malloc(sizeof(float) * input_array_size);

    int* d_input_array_size;
    float* d_input_array1; 
    float* d_input_array2; 
    float* d_output_array;


    const int thread_count = min(input_array_size/ OPERATIONS_PER_THREAD + 1, 1024);
    const int block_count = input_array_size / OPERATIONS_PER_THREAD / 1024 + 1;


    hipMalloc((void**) &d_input_array1, sizeof(float) * input_array_size);
    hipMalloc((void**) &d_input_array2, sizeof(float) * input_array_size);
    hipMalloc((void**) &d_output_array, sizeof(float) * input_array_size);
    hipMalloc((void**) &d_input_array_size, sizeof(int));

    hipMemcpy(d_input_array1, h_input_array1, sizeof(float) * input_array_size, hipMemcpyHostToDevice);
    hipMemcpy(d_input_array2, h_input_array2, sizeof(float) * input_array_size, hipMemcpyHostToDevice);
    hipMemcpy(d_input_array_size, h_input_array_size, sizeof(int), hipMemcpyHostToDevice);
    
    auto program_start_time = std::chrono::high_resolution_clock::now();

    vector_addition_kernel <<<block_count, thread_count>>> (d_input_array1, d_input_array2, d_output_array, d_input_array_size);
    hipDeviceSynchronize();

    auto program_end_time = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_output_array, d_output_array, sizeof(float) * output_array_size, hipMemcpyDeviceToHost);

    hipFree(d_input_array_size);
    hipFree(d_input_array1);      
    hipFree(d_input_array2);        
    hipFree(d_output_array);  


    std::vector<float> result = c_array_to_std_vector(h_output_array, output_array_size);

    
    float program_duration = std::chrono::duration_cast<std::chrono::microseconds>(program_end_time - program_start_time).count();
    std::cout << "Time (microseconds) in cuda vector addition program: " << program_duration << std::endl;

    return result;
}
