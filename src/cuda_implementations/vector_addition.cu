#include "hip/hip_runtime.h"
#include "cuda_matrix_lib.h"

#define OPERATIONS_PER_THREAD 1024


__global__ void vector_addition_kernel(float* d_vector1, float* d_vector2, float* d_output_vector, int* vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int thread_start_index = thread_id * OPERATIONS_PER_THREAD;

    for (int index = 0; index < OPERATIONS_PER_THREAD; index++) {
        int input_vector_index = thread_start_index + index;

        if (input_vector_index >= *vector_size) return;

        d_output_vector[input_vector_index] = d_vector1[input_vector_index] + d_vector2[input_vector_index];
    }

}


std::vector<float> CudaMatrixLib::vector_addition(std::vector<float> vector1, std::vector<float> vector2) {

    if (vector1.size() != vector2.size()) {
        return std::vector<float>();
    }
    

    int vector_size = vector1.size();
    int* h_vector_size = &vector_size;
    int* d_vector_size;

    float* h_vector1 = &vector1[0]; float* h_vector2 = &vector2[0];
    float* h_output_vector = (float*) malloc(sizeof(float) * vector_size);

    float* d_vector1; float* d_vector2; float* d_output_vector;


    const int thread_count = min(vector_size/ OPERATIONS_PER_THREAD + 1, 1024);
    const int block_count = vector_size / OPERATIONS_PER_THREAD / 1024 + 1;


    hipMalloc((void**) &d_vector1, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector2, sizeof(float) * vector_size);
    hipMalloc((void**) &d_output_vector, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector_size, sizeof(int));

    hipMemcpy(d_vector1, h_vector1, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector2, h_vector2, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_size, h_vector_size, sizeof(int), hipMemcpyHostToDevice);
    
    vector_addition_kernel <<<block_count, thread_count>>> (d_vector1, d_vector2, d_output_vector, d_vector_size);

    hipMemcpy(h_output_vector, d_output_vector, sizeof(float) * vector_size, hipMemcpyDeviceToHost);

    hipFree(d_vector1);      
    hipFree(d_vector2);        
    hipFree(d_output_vector);  
    hipFree(d_vector_size);


    std::vector<float> result;
    result.insert(result.end(), h_output_vector, h_output_vector + vector_size); 
    return result;
}
