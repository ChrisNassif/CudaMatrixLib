#include "hip/hip_runtime.h"
#include "cuda_matrix_lib.h"

__global__ void vector_addition_kernel(float* d_vector1, float* d_vector2, float* d_output_vector, int* vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    if (thread_id >= *vector_size) return;

    d_output_vector[thread_id] = d_vector1[thread_id] + d_vector2[thread_id];

}


std::vector<float> CudaMatrixLib::vector_addition(std::vector<float> vector1, std::vector<float> vector2) {

    if (vector1.size() != vector2.size()) {
        return std::vector<float>();
    }
    

    int vector_size = vector1.size();
    int* h_vector_size = &vector_size;
    int* d_vector_size;

    float* h_vector1 = &vector1[0]; float* h_vector2 = &vector2[0];
    float* h_output_vector = (float*) malloc(sizeof(float) * vector_size);

    float* d_vector1; float* d_vector2; float* d_output_vector;


    const int threadCount = min(vector_size, 1024);
    const int blockCount = vector_size / 1024 + 1;


    hipMalloc((void**) &d_vector1, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector2, sizeof(float) * vector_size);
    hipMalloc((void**) &d_output_vector, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector_size, sizeof(int));

    hipMemcpy(d_vector1, h_vector1, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector2, h_vector2, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_size, h_vector_size, sizeof(int), hipMemcpyHostToDevice);
    
    vector_addition_kernel <<<blockCount, threadCount>>> (d_vector1, d_vector2, d_output_vector, d_vector_size);

    hipMemcpy(h_output_vector, d_output_vector, sizeof(float) * vector_size, hipMemcpyDeviceToHost);

    hipFree(d_vector1);      
    hipFree(d_vector2);        
    hipFree(d_output_vector);  
    hipFree(d_vector_size);


    std::vector<float> result;
    result.insert(result.end(), h_output_vector, h_output_vector + vector_size); 
    return result;
}
