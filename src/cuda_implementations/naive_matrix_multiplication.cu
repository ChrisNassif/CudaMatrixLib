#include "cuda_matrix_lib.h"
#include "cuda_kernels.h"


Matrix CudaMatrixLib::naive_matrix_multiplication(Matrix input_matrix1, Matrix input_matrix2) {    
    if (input_matrix1.size() == 0 || input_matrix2.size() == 0) {
        return Matrix();
    }

    int input_matrix1_number_of_rows = input_matrix1.size();
    int input_matrix1_number_of_columns = input_matrix1[0].size();

    int input_matrix2_number_of_rows = input_matrix2.size();
    int input_matrix2_number_of_columns = input_matrix2[0].size();

    if (input_matrix1_number_of_columns != input_matrix2_number_of_rows) {
        return Matrix();
    }

    int output_matrix_number_of_rows = input_matrix1_number_of_rows;
    int output_matrix_number_of_columns = input_matrix2_number_of_columns;
    
    float* h_input_matrix1 = (float*) malloc(sizeof(float) * input_matrix1_number_of_rows * input_matrix1_number_of_columns);
    float* h_input_matrix2 = (float*) malloc(sizeof(float) * input_matrix2_number_of_rows * input_matrix2_number_of_columns);
    float* h_output_matrix = (float*) malloc(sizeof(float) * output_matrix_number_of_rows * output_matrix_number_of_columns);
    int* h_input_matrix1_number_of_rows_M = &(input_matrix1_number_of_rows);
    int* h_input_matrix1_number_of_columns_K = &(input_matrix1_number_of_columns);
    int* h_input_matrix2_number_of_columns_N = &(input_matrix2_number_of_columns);

    matrix_to_c_array(input_matrix1, h_input_matrix1);
    matrix_to_c_array(input_matrix2, h_input_matrix2);

    // print_float_array(h_input_c_matrix1, input_matrix1_number_of_rows * input_matrix1_number_of_columns);
    // print_float_array(h_input_c_matrix2, input_matrix2_number_of_rows * input_matrix2_number_of_columns);
    
    float* d_input_matrix1; 
    float* d_input_matrix2;
    float* d_output_matrix;
    int* d_input_matrix1_number_of_rows_M;
    int* d_input_matrix1_number_of_columns_K;
    int* d_input_matrix2_number_of_columns_N;


    hipMalloc((void**) &d_input_matrix1, sizeof(float) * input_matrix1_number_of_rows * input_matrix1_number_of_columns);
    hipMalloc((void**) &d_input_matrix2, sizeof(float) * input_matrix2_number_of_rows * input_matrix2_number_of_columns);
    hipMalloc((void**) &d_output_matrix, sizeof(float) * output_matrix_number_of_rows * output_matrix_number_of_columns);
    hipMalloc((void**) &d_input_matrix1_number_of_rows_M, sizeof(int));
    hipMalloc((void**) &d_input_matrix1_number_of_columns_K, sizeof(int));
    hipMalloc((void**) &d_input_matrix2_number_of_columns_N, sizeof(int));

    hipMemcpy(d_input_matrix1, h_input_matrix1, sizeof(float) * input_matrix1_number_of_rows * input_matrix1_number_of_columns, hipMemcpyHostToDevice);
    hipMemcpy(d_input_matrix2, h_input_matrix2, sizeof(float) * input_matrix2_number_of_rows * input_matrix2_number_of_columns, hipMemcpyHostToDevice);
    hipMemcpy(d_output_matrix, h_output_matrix, sizeof(float) * output_matrix_number_of_rows * output_matrix_number_of_columns, hipMemcpyHostToDevice);
    hipMemcpy(d_input_matrix1_number_of_rows_M, h_input_matrix1_number_of_rows_M, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_input_matrix1_number_of_columns_K, h_input_matrix1_number_of_columns_K, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_input_matrix2_number_of_columns_N, h_input_matrix2_number_of_columns_N, sizeof(int), hipMemcpyHostToDevice);

    // TODO: CHANGE THIS
    const auto thread_count = dim3(16, 16);
    const auto block_count = 1;

    auto program_start_time = std::chrono::high_resolution_clock::now();

    naive_matrix_multiplication_kernel <<<block_count, thread_count>>> (
        d_input_matrix1, d_input_matrix2, d_output_matrix,
        d_input_matrix1_number_of_rows_M, d_input_matrix1_number_of_columns_K, d_input_matrix2_number_of_columns_N
    );
    hipDeviceSynchronize();

    auto program_end_time = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_output_matrix, d_output_matrix, sizeof(float) * output_matrix_number_of_rows * output_matrix_number_of_columns, hipMemcpyDeviceToHost);

    hipFree(d_input_matrix1);
    hipFree(d_input_matrix2);
    hipFree(d_output_matrix);
    hipFree(d_input_matrix1_number_of_rows_M);  
    hipFree(d_input_matrix1_number_of_columns_K);
    hipFree(d_input_matrix2_number_of_columns_N);

    free(h_input_matrix1);
    free(h_input_matrix2);

    print_float_array(h_output_matrix, input_matrix1_number_of_rows * input_matrix2_number_of_columns);

    Matrix result = c_array_to_matrix(h_output_matrix, output_matrix_number_of_rows, output_matrix_number_of_columns);

    free(h_output_matrix);
    
    float program_duration = std::chrono::duration_cast<std::chrono::microseconds>(program_end_time - program_start_time).count();
    std::cout << "Time (microseconds) in cuda scalar multiplication program: " << program_duration << std::endl;

    return result;
}
