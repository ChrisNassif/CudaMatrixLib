#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_matrix_lib.h"
#include <chrono>


__global__ void hadamard_product_kernel(float* d_vector1, float* d_vector2, float* d_output_vector, int* vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    if (thread_id >= *vector_size) return;

    d_output_vector[thread_id] = d_vector1[thread_id] * d_vector2[thread_id];

}


std::vector<float> hadamard_product(std::vector<float> vector1, std::vector<float> vector2) {

    auto program_start_time = std::chrono::high_resolution_clock::now();

    if (vector1.size() != vector2.size()) {
        return std::vector<float>();
    }
    
    int vector_size = vector1.size();
    int* h_vector_size = &vector_size;
    int* d_vector_size;

    float* h_vector1 = &vector1[0]; float* h_vector2 = &vector2[0];
    float* h_output_vector = (float*) malloc(sizeof(float) * vector_size);

    float* d_vector1; float* d_vector2; float* d_output_vector;


    const int threadCount = min(vector_size, 1024);
    const int blockCount = vector_size / 1024 + 1;


    hipMalloc((void**) &d_vector1, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector2, sizeof(float) * vector_size);
    hipMalloc((void**) &d_output_vector, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector_size, sizeof(int));

    hipMemcpy(d_vector1, h_vector1, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector2, h_vector2, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_size, h_vector_size, sizeof(int), hipMemcpyHostToDevice);
    
    hadamard_product_kernel <<<blockCount, threadCount>>> (d_vector1, d_vector2, d_output_vector, d_vector_size);

    hipMemcpy(h_output_vector, d_output_vector, sizeof(float) * vector_size, hipMemcpyDeviceToHost);

    hipFree(d_vector1);      
    hipFree(d_vector2);        
    hipFree(d_output_vector);  
    hipFree(d_vector_size);

    std::vector<float> result;
    result.insert(result.end(), h_output_vector, h_output_vector + vector_size);

    
    auto program_end_time = std::chrono::high_resolution_clock::now();
    float program_duration = std::chrono::duration_cast<std::chrono::microseconds>(program_end_time - program_start_time).count();
    std::cout << "Time (microseconds) in cuda hadamard product program: " << program_duration << std::endl;

    return result;
}

