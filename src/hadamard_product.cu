#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_matrix_lib.h"
#include <chrono>


__global__ void hadamard_product_kernel(float* d_vector1, float* d_vector2, float* d_output_vector, int* vector_size) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    if (thread_id >= *vector_size) return;

    d_output_vector[thread_id] = d_vector1[thread_id] * d_vector2[thread_id];

}


std::vector<float> hadamard_product(std::vector<float> vector1, std::vector<float> vector2) {

    if (vector1.size() != vector2.size()) {
        return std::vector<float>();
    }
    

    int vector_size = vector1.size();
    int* h_vector_size = &vector_size;
    int* d_vector_size;

    float* h_vector1 = &vector1[0]; float* h_vector2 = &vector2[0];
    float* h_output_vector = (float*) malloc(sizeof(float) * vector_size);

    float* d_vector1; float* d_vector2; float* d_output_vector;


    const int threadCount = min(vector_size, 1024);
    const int blockCount = vector_size / 1024 + 1;


    hipMalloc((void**) &d_vector1, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector2, sizeof(float) * vector_size);
    hipMalloc((void**) &d_output_vector, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector_size, sizeof(int));

    hipMemcpy(d_vector1, h_vector1, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector2, h_vector2, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector_size, h_vector_size, sizeof(int), hipMemcpyHostToDevice);
    
    hadamard_product_kernel <<<blockCount, threadCount>>> (d_vector1, d_vector2, d_output_vector, d_vector_size);

    hipMemcpy(h_output_vector, d_output_vector, sizeof(float) * vector_size, hipMemcpyDeviceToHost);

    hipFree(d_vector1);      
    hipFree(d_vector2);        
    hipFree(d_output_vector);  
    hipFree(d_vector_size);


    std::vector<float> result;
    result.insert(result.end(), h_output_vector, h_output_vector + vector_size); 
    return result;
    return std::vector<float>(10, 0);
}




// int main( void ) {

//     int vector_size = 100000;
//     std::vector<float> vector1(vector_size, 0);
//     std::vector<float> vector2(vector_size, 0);

//     int index;
//     for (index = 0; index < vector_size; index++) {
//         vector1[index] = (float)(rand() % 10);
//         vector2[index] = (float)(rand() % 10);
//     }
    
//     print_vector(hadamard_product(vector1, vector2));

//     // return 0;
// }
