#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>    
#include <vector> 



__global__ void dot_product_kernel(float* d_vector1, float* d_vector2, float* d_output_vector) {
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    d_output_vector[thread_id] = d_vector1[thread_id] * d_vector2[thread_id];
}

std::vector<float> dot_product(std::vector<float> vector1, std::vector<float> vector2) {

    if (vector1.size() != vector2.size() || vector1.size() >= 1024) {
        return std::vector<float>();
    }

    const int vector_size = vector1.size();

    const int threadCount = vector_size;
    const int blockCount = 1;

    float* h_vector1 = &vector1[0]; float* h_vector2 = &vector2[0];
    float* h_output_vector = (float*) malloc(sizeof(float) * vector_size);

    float* d_vector1; float* d_vector2; float* d_output_vector;


    hipMalloc((void**) &d_vector1, sizeof(float) * vector_size);
    hipMalloc((void**) &d_vector2, sizeof(float) * vector_size);
    hipMalloc((void**) &d_output_vector, sizeof(float) * vector_size);

    hipMemcpy(d_vector1, h_vector1, sizeof(float) * vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector2, h_vector2, sizeof(float) * vector_size, hipMemcpyHostToDevice);

    dot_product_kernel <<<blockCount, threadCount>>> (d_vector1, d_vector2, d_output_vector);

    hipMemcpy(h_vector1, d_vector1, sizeof(float) * vector_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_vector2, d_vector2, sizeof(float) * vector_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_output_vector, d_output_vector, sizeof(float) * vector_size, hipMemcpyDeviceToHost);

    hipFree(d_vector1);
    hipFree(d_vector2);
    hipFree(d_output_vector);


    std::vector<float> result;
    result.insert(result.end(), h_output_vector, h_output_vector + vector_size); 
    return result;
}

void print_vector(std::vector<float> vector) {

    printf("[");
    int index;
    for (index = 0; index < vector.size(); index++) {
        if (index == vector.size() - 1) {
            printf("%f]\n", vector[index]);
            continue;
        }
        printf("%f, ", vector[index]);
    }
}

int main( void ) {

    // float vector1[] = (float*) malloc(sizeof(float) * 10);
    // float vector2[] = (float*) malloc(sizeof(float) * 10);

    std::vector<float> vector1 = {0, 1, 3, 1, 4, 2};
    std::vector<float> vector2 = {1, 4, 1, 2, 3, 1};

    // int index;
    // for (index = 0; index < 10; index++) {
    //     vector1[index] = rand() % 10;
    //     vector2[index] = rand() % 10;
    // }

    print_vector(dot_product(vector1, vector2));

    return 0;
}
